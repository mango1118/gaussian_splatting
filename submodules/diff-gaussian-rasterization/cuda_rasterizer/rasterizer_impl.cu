#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"   // 自定义的光栅化实现相关头文件
#include <iostream>             // 用于输入输出流（如打印日志）
#include <fstream>              // 用于文件操作
#include <algorithm>            // 用于常见的算法操作（如排序）
#include <numeric>              // 用于数字计算（如累加）
#include <hip/hip_runtime.h>               // CUDA API
#include "hip/hip_runtime.h"       // CUDA运行时API
#include "" // 用于设备的启动参数
#include <hipcub/hipcub.hpp>          // CUB库：CUDA高级算法库
#include <cub/device/device_radix_sort.cuh> // CUB设备排序算法（基数排序）
#define GLM_FORCE_CUDA          // 强制启用CUDA支持的GLM（数学库）
#include <glm/glm.hpp>          // GLM：OpenGL数学库

#include <hip/hip_cooperative_groups.h> // 协同组（cooperative groups）头文件，用于高效的线程协同计算
#include <cooperative_groups/reduce.h> // 协同组的归约（reduction）操作
namespace cg = cooperative_groups;  // 简化命名空间为cg

#include "auxiliary.h"          // 辅助函数的头文件
#include "forward.h"            // 向前传播相关的头文件
#include "backward.h"           // 向后传播相关的头文件


// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
    uint32_t msb = sizeof(n) * 4;  // 计算32位整数的位数（通常为32）
    uint32_t step = msb;           // 步长初始化为msb的值
    while (step > 1)               // 当步长大于1时，继续调整
    {
        step /= 2;                  // 每次将步长减半
        if (n >> msb)               // 检查n的高位是否为1
            msb += step;            // 如果是，msb增加步长
        else
            msb -= step;            // 否则，msb减小步长
    }
    if (n >> msb)                   // 最终检查高位的值
        msb++;                       // 如果是1，则msb增加
    return msb;
}


// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
    const float* orig_points,
    const float* viewmatrix,
    const float* projmatrix,
    bool* present)
{
    // 获取当前线程的索引
    auto idx = cg::this_grid().thread_rank();
    
    // 如果线程的索引超出了需要处理的点数量P，则返回
    if (idx >= P)
        return;

    float3 p_view;
    
    // 使用辅助函数in_frustum进行视锥体内检测，p_view是检测过程中使用的视图坐标
    present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}


// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
    int P,
    const float2* points_xy,
    const float* depths,
    const uint32_t* offsets,
    uint64_t* gaussian_keys_unsorted,
    uint32_t* gaussian_values_unsorted,
    int* radii,
    dim3 grid)
{
    // 获取当前线程在网格中的索引
    auto idx = cg::this_grid().thread_rank();
    
    // 如果当前线程的索引超出点的总数P，则返回
    if (idx >= P)
        return;

    // 如果高斯的半径大于0，说明它是可见的
    if (radii[idx] > 0)
    {
        // 查找当前高斯的偏移量，用于写入键值对
        uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
        uint2 rect_min, rect_max;

        // 获取该高斯所在的矩形区域的最小和最大坐标
        getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

        // 遍历该高斯的矩形区域，检查它与哪些瓦片有重叠
        // 对于每个与该高斯重叠的瓦片，生成一个键值对
        // 键是 | tile ID | depth |，值是高斯的 ID
        for (int y = rect_min.y; y < rect_max.y; y++)
        {
            for (int x = rect_min.x; x < rect_max.x; x++)
            {
                // 计算当前瓦片的 ID
                uint64_t key = y * grid.x + x;
                key <<= 32;  // 将瓦片 ID 移到高位
                key |= *((uint32_t*)&depths[idx]);  // 将该高斯的深度值放入低位

                // 存储键值对
                gaussian_keys_unsorted[off] = key;
                gaussian_values_unsorted[off] = idx;

                // 更新偏移量
                off++;
            }
        }
    }
}


// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
    // 获取当前线程在网格中的索引
    auto idx = cg::this_grid().thread_rank();
    
    // 如果当前线程的索引大于高斯点总数L，则跳过当前线程
    if (idx >= L)
        return;

    // 从键中读取瓦片 ID，键的高位存储的是瓦片 ID，低位是深度信息
    uint64_t key = point_list_keys[idx];
    uint32_t currtile = key >> 32;  // 提取当前高斯所在瓦片的 ID

    // 如果是第一个元素，设置当前瓦片的开始位置
    if (idx == 0)
        ranges[currtile].x = 0;
    else
    {
        // 获取前一个高斯所在瓦片的 ID
        uint32_t prevtile = point_list_keys[idx - 1] >> 32;
        
        // 如果当前瓦片与前一个瓦片不同，更新前一个瓦片的结束位置
        if (currtile != prevtile)
        {
            ranges[prevtile].y = idx;  // 更新前一个瓦片的结束位置
            ranges[currtile].x = idx;  // 设置当前瓦片的开始位置
        }
    }

    // 如果是最后一个元素，设置当前瓦片的结束位置
    if (idx == L - 1)
        ranges[currtile].y = L;
}


// Mark Gaussians as visible/invisible, based on view frustum testing
// 通过视锥体测试标记哪些高斯点是可见的
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

// 从chunk缓冲区提取和初始化GeometryState结构
CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color,
	float* depth,
	bool antialiasing,
	int* radii,
	bool debug)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	size_t chunk_size = required<GeometryState>(P);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	size_t img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,
		geomState.means2D,
		geomState.depths,
		geomState.cov3D,
		geomState.rgb,
		geomState.conic_opacity,
		tile_grid,
		geomState.tiles_touched,
		prefiltered,
		antialiasing
	), debug)

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid)
	CHECK_CUDA(, debug)

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)

	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)

	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	CHECK_CUDA(FORWARD::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		geomState.means2D,
		feature_ptr,
		geomState.conic_opacity,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		out_color,
		geomState.depths,
		depth), debug)

	return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	const float* dL_invdepths,
	float* dL_dmean2D,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dinvdepth,
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot,
	bool antialiasing,
	bool debug)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		geomState.depths,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		dL_invdepths,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor,
		dL_dinvdepth), debug);

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		opacities,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		dL_dinvdepth,
		dL_dopacity,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot,
		antialiasing), debug);
}
